#include "hip/hip_runtime.h"
/* Short explanation on the way to add fields and terms in this file
 *
 * evolver is a class that merely calls updates on all fields and terms
 * the arguments on its constructor are 
 *
 *      evolver system(x,           sx,             sy,             dx,       dy,       dt);
 *                     Use CUDA | x-system size | y-system size | delta_x | delta_y | delta_t
 *
 * To this evolver we can add fields:
 *
 *      system.createField( name, dynamic );
 *
 * name is a string and dynamic if a boolean that sets whether the field
 * is set in each step through a time derivative or through an equality.
 *
 * To each field we can add terms
 *      
 *      system.createTerm(  field_name, prefactor, {field_1, ..., field_n}  );
 *
 *  This term would be a term of "field_name", with that prefactor, that multiplies
 *  fields field_1 to field_n.
 */ 

#include <cmath>
#include <cstdlib>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <iostream>
#include <ostream>
#include "../../inc/defines.h"
#include "../../inc/evolver.h"
#include "../../inc/field.h"
#include "../../inc/term.h"
#include "parse_input.h"

#ifdef WITHCUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#endif


void zero_boundaries_y(float2*, int, int);
__global__ void zero_k_y(float2*, int, int);

void phi_boundaries_y(float2*, int, int);
__global__ void phi_k_y(float2*, int, int);

int main (int argc, char* argv[]) 
{
    bool use_GPU = true; 

    if (argc < 2) 
    {
    printf("Usage: model param_file level\n");
    return 1;
    }

    system_parameters sys_par= parser(argv[1]);

    evolver system(use_GPU, sys_par.Nx, sys_par.Ny, sys_par.dx, sys_par.dy, sys_par.dt, sys_par.NSave);

    system.createField("iqxQxx", false);// 0
    system.createField("iqyQxx", false);// 1
    system.createField("iqxQxy", false);// 2
    system.createField("iqyQxy", false);// 3
    system.createField("iqxphi", false);// 4
    system.createField("iqyphi", false);// 5
    system.createField("sigxx", false); // 6
    system.createField("sigxy", false); // 7
    system.createField("vx", false);    // 8
    system.createField("vy", false);    // 9
    system.createField("wxy", false);   // 10
    system.createField("Q2", false);    // 11
    system.createField("Qxx", true);    // 12
    system.createField("Qxy", true);    // 13
    system.createField("phi", true);    // 14

    system.createField("ident", false);    // 15
    system.createField("FgY", false);    // 16
    // system.createField("P_g", false);    // 17
    // system.createField("P_c", false);    // 17

    // CONSTANTS
    // v and Q
    float gamma_fric = sys_par.GammaFric;
    float eta = sys_par.eta;
    float aQ = sys_par.aQ;
    float bQ = sys_par.bQ;
    float kQ = sys_par.KQ;
    float lambda = sys_par.lambda;
    float gamma = sys_par.gammaQ;
    float alpha = sys_par.alpha;
    // phi
    float a = sys_par.aPhi;
    float b = sys_par.bPhi;
    float M = sys_par.MPhi;
    float phi0 = std::sqrt(-a/b);
    float k = sys_par.kappaPhi;
    float ka = sys_par.kappaHatPhi;


    system.fields[8]->hasBC = true;
    system.fields[8]->boundary = zero_boundaries_y;        
    system.fields[9]->hasBC = true;
    system.fields[9]->boundary = zero_boundaries_y;        
  
    system.fields[14]->hasBC = true;
    system.fields[14]->boundary = phi_boundaries_y;        

    // Implicit terms
    system.fields[8]->implicit.push_back({eta, 1, 0, 0, 0});
    system.fields[8]->implicit.push_back({gamma_fric, 0, 0, 0, 0});
    system.fields[9]->implicit.push_back({eta, 1, 0, 0, 0});
    system.fields[9]->implicit.push_back({gamma_fric, 0, 0, 0, 0});
    system.fields[12]->implicit.push_back({-aQ/gamma});
    system.fields[12]->implicit.push_back({-kQ/gamma, 1, 0, 0, 0});
    system.fields[13]->implicit.push_back({-aQ/gamma});
    system.fields[13]->implicit.push_back({-kQ/gamma, 1, 0, 0, 0});
    system.fields[14]->implicit.push_back({-M*a, 1, 0, 0, 0});
    system.fields[14]->implicit.push_back({-M*k, 2, 0, 0, 0});

    //Explicit terms
    system.createTerm("iqxQxx", {{1.0f, 0, 1, 0, 0}}, {"Qxx"});
    system.createTerm("iqyQxx", {{1.0f, 0, 0, 1, 0}}, {"Qxx"});
    system.createTerm("iqxQxy", {{1.0f, 0, 1, 0, 0}}, {"Qxy"});
    system.createTerm("iqyQxy", {{1.0f, 0, 0, 1, 0}}, {"Qxy"});
    system.createTerm("iqxphi", {{1.0f, 0, 1, 0, 0}}, {"phi"});
    system.createTerm("iqyphi", {{1.0f, 0, 0, 1, 0}}, {"phi"});

    system.createTerm("sigxx", {{alpha/2.0f}}, {"Qxx"});
    system.createTerm("sigxy", {{alpha/2.0f}}, {"Qxy"});
    system.createTerm("sigxx", {{alpha/(2.0f*phi0)}}, {"phi", "Qxx"});
    system.createTerm("sigxy", {{alpha/(2.0f*phi0)}}, {"phi", "Qxy"});
    system.createTerm("sigxx", {{-ka/2.0f}}, {"iqxphi", "iqxphi"});
    system.createTerm("sigxx", {{ka/2.0f}}, {"iqyphi", "iqyphi"});
    system.createTerm("sigxy", {{-ka}}, {"iqxphi", "iqyphi"});

    // alc backflow
    system.createTerm("sigxx", {{lambda*aQ}},{"Qxx"});
    system.createTerm("sigxx", {{lambda*bQ}},{"Q2", "Qxx"});
    system.createTerm("sigxx", {{lambda*kQ, 1, 0, 0, 0}},{"Qxx"});
    system.createTerm("sigxy", {{lambda*aQ}},{"Qxy"});
    system.createTerm("sigxy", {{lambda*bQ}},{"Q2", "Qxy"});
    system.createTerm("sigxy", {{lambda*kQ, 1, 0, 0, 0}},{"Qxy"});

    system.createTerm("Qxx", {{lambda, 0, 1, 0, 0}}, {"vx"});
    system.createTerm("Qxx", {{-2.0f}}, {"Qxy", "wxy"});
    system.createTerm("Qxx", {{-bQ/gamma}}, {"Q2", "Qxx"});
    system.createTerm("Qxx", {{-1.0f}}, {"vx", "iqxQxx"});
    system.createTerm("Qxx", {{-1.0f}}, {"vy", "iqyQxx"});

    system.createTerm("Qxy", {{lambda/2, 0, 1, 0, 0}}, {"vy"});
    system.createTerm("Qxy", {{lambda/2, 0, 0, 1, 0}}, {"vx"});
    system.createTerm("Qxy", {{2.0f}}, {"Qxx", "wxy"});
    system.createTerm("Qxy", {{-bQ/gamma}}, {"Q2", "Qxy"});
    system.createTerm("Qxy", {{-1.0f}}, {"vx", "iqxQxy"});
    system.createTerm("Qxy", {{-1.0f}}, {"vy", "iqyQxy"});

    system.createTerm("wxy", {{0.5f, 0, 1, 0, 0}}, {"vy"});
    system.createTerm("wxy", {{-0.5f, 0, 0, 1, 0}}, {"vx"});

    system.createTerm("Q2", {{1.0f}}, {"Qxx", "Qxx"});
    system.createTerm("Q2", {{1.0f}}, {"Qxy", "Qxy"});

    system.createTerm("phi", {{-M*b, 1, 0, 0, 0}}, {"phi", "phi", "phi"});
    system.createTerm("phi", {{-1.0f}}, {"vx", "iqxphi"});
    system.createTerm("phi", {{-1.0f}}, {"vy", "iqyphi"});

    // Terms for vx and vy
    pres iqx = {1.0f, 0, 1, 0, 0};
    pres iqy = {1.0f, 0, 0, 1, 0};
    pres miqy = {-1.0f, 0, 0, 1, 0};
    pres miqy3 = {-1.0f, 0, 0, 3, 2};
    pres iqx3 = {1.0f, 0, 3, 0, 2};
    pres iqx2iqy = {1.0f, 0, 2, 1, 2};
    pres miqxiqy2 = {-1.0f, 0, 1, 2, 2};
    pres iqxiqy2 = {1.0f, 0, 1, 2, 2};
    system.createTerm("vx", {iqx, iqx3, miqxiqy2}, {"sigxx"});
    system.createTerm("vx", {iqy, iqx2iqy, iqx2iqy}, {"sigxy"});
    system.createTerm("vy", {miqy, miqy3, iqx2iqy}, {"sigxx"});
    system.createTerm("vy", {iqx, iqxiqy2, iqxiqy2}, {"sigxy"});

    // Gravitational Terms
    float g = sys_par.gGrav;
    float rho_1 = 2.0f;
    float rho_2 = 1.0f;
   
    system.createTerm("FgY", {{-(g*(rho_1-rho_2)/(2*phi0))}}, {"phi"});
    system.createTerm("FgY", {{-(g*(rho_1-rho_2)/2)}}, {"ident"});
    system.createTerm("vx", {{1.0f, 0, 1, 1, 2}}, {"FgY"});
    system.createTerm("vy", {{-1.0f, 0, 2, 0, 2}}, {"FgY"});

    switch (sys_par.initialConfig)
    {
    case 1:
        // Flat interface initial
        std::srand(sys_par.seed);
        for (int i = 0; i < sys_par.Nx; i++)
        {
            for (int j = 0; j < sys_par.Ny; j++)
            {
                system.fields[14]->real_array[j*sys_par.Nx+i].x = -std::tanh(j-sys_par.Ny/2) + sys_par.phi0_noise * 0.01f * (float)(std::rand() % 200 - 100);;
                system.fields[14]->real_array[j*sys_par.Nx+i].y = 0.0f;

                system.fields[12]->real_array[i*sys_par.Nx+j].x = sys_par.Qxx0_noise * 0.01f * (float)(std::rand() % 200 - 100);
                system.fields[12]->real_array[i*sys_par.Nx+j].y = 0.0f;
                system.fields[13]->real_array[i*sys_par.Nx+j].x = sys_par.Qxy0_noise * 0.01f * (float)(std::rand() % 200 - 100);
                system.fields[13]->real_array[i*sys_par.Nx+j].y = 0.0f;
            }
        }

    break;
    }


    for (int i = 0; i < sys_par.Nx; i++)
    {
        for (int j = 0; j < sys_par.Ny; j++)
        {
            system.fields[15]->real_array[j*sys_par.Nx+i].x =  1.0f;
            system.fields[15]->real_array[j*sys_par.Nx+i].y =  0.0f;
        }
    }


    hipMemcpy(system.fields[12]->real_array_d, system.fields[12]->real_array, sys_par.Nx*sys_par.Ny*sizeof(float2), hipMemcpyHostToDevice);
    hipMemcpy(system.fields[12]->comp_array_d, system.fields[12]->comp_array, sys_par.Nx*sys_par.Ny*sizeof(float2), hipMemcpyHostToDevice);
    hipMemcpy(system.fields[13]->real_array_d, system.fields[13]->real_array, sys_par.Nx*sys_par.Ny*sizeof(float2), hipMemcpyHostToDevice);
    hipMemcpy(system.fields[13]->comp_array_d, system.fields[13]->comp_array, sys_par.Nx*sys_par.Ny*sizeof(float2), hipMemcpyHostToDevice);
    hipMemcpy(system.fields[14]->real_array_d, system.fields[14]->real_array, sys_par.Nx*sys_par.Ny*sizeof(float2), hipMemcpyHostToDevice);
    hipMemcpy(system.fields[14]->comp_array_d, system.fields[14]->comp_array, sys_par.Nx*sys_par.Ny*sizeof(float2), hipMemcpyHostToDevice);
    hipMemcpy(system.fields[15]->real_array_d, system.fields[15]->real_array, sys_par.Nx*sys_par.Ny*sizeof(float2), hipMemcpyHostToDevice);
    hipMemcpy(system.fields[15]->comp_array_d, system.fields[15]->comp_array, sys_par.Nx*sys_par.Ny*sizeof(float2), hipMemcpyHostToDevice);
    system.fields[12]->toComp();
    system.fields[13]->toComp();
    system.fields[14]->toComp();
    system.fields[15]->toComp();


    switch (sys_par.noiseConfig)
    {
    case 1:
    //Conserved Phase Field Noise
        system.fields[14]->isNoisy = true;
        system.fields[14]->noiseType = GaussianWhite;
        system.fields[14]->noise_amplitude = {sys_par.phi_D,1,0,0,0};
        break;
    
    default:
    // No noise in the dynamics
        break;
    }

    
    for (int i = 0; i < system.fields.size(); i++)
    {
        system.fields[i]->prepareDevice();
        system.fields[i]->precalculateImplicit(system.dt);
        system.fields[i]->outputToFile = false;
    }
    // system.fields[11]->outputToFile = true;
    // system.fields[12]->outputToFile = true;
    // system.fields[13]->outputToFile = true;
    system.fields[14]->outputToFile = true;

    int steps = sys_par.NSteps;
    int check = steps/100;
    if (check < 1) check = 1;
    
    if (argv[2] == "v" )
    {
        system.printInformation();
    }


    for (int i = 0; i < steps; i++)
    {
        system.advanceTime();
        if (i % check == 0)
        {
            std::cout << "Progress: " << i/check << "%\r";
            std::cout.flush();
        }
    }

    return 0;
}


void zero_boundaries_y(float2 *real_array, int sx, int sy)
{
    dim3 TPB(32,32);
    dim3 blocks((sx+31)/32, (sy+31)/32);
    zero_k_y<<<blocks, TPB>>>(real_array, sx, sy);
}

__global__ void zero_k_y(float2 *real_array, int sx, int sy)
{
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        int j = blockIdx.y * blockDim.y + threadIdx.y;
        int index = j*sx+i;

        if (index < sx*sy )
        {
            if (j < sy/8 || j > sy - sy/8)
                real_array[index].x = 0.0f;
        }
}


void phi_boundaries_y(float2 *real_array, int sx, int sy)
{
    dim3 TPB(32,32);
    dim3 blocks((sx+31)/32, (sy+31)/32);
    phi_k_y<<<blocks, TPB>>>(real_array, sx, sy);
}

__global__ void phi_k_y(float2 *real_array, int sx, int sy)
{
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        int j = blockIdx.y * blockDim.y + threadIdx.y;
        int index = j*sx+i;

        if (index < sx*sy )
        {
            if (j < sy/8)
            {
                real_array[index].x = 1.0f;
            }

            if (j > sy - sy/8)
            {
                real_array[index].x = -1.0f;
            }
        }
}

